#include "hip/hip_runtime.h"
#include "utils.h"
#include <stdio.h>
#include <math.h>       /* ceil */

// Max Threads per block in GeForce 210
#define TxB 512

__global__
void rgba_to_grey_kernel(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
  
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  uchar4 px = rgbaImage[i]; // thread pixel to process
  greyImage[i] = .299f * px.x +
                 .587f * px.y +
                 .114f * px.z;
}

void rgba_to_grey(uchar4 * const d_rgbaImage,
                  unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{

  
  long long int total_px = numRows * numCols;  // total pixels
  long int grids_n = ceil(total_px / TxB); // grids numer
  const dim3 blockSize(TxB, 1, 1);
  const dim3 gridSize(grids_n, 1, 1);
  rgba_to_grey_kernel<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}
