#include "hip/hip_runtime.h"
#include "utils.h"
#include <stdio.h>
#include <math.h>       /* ceil */

// Max Threads per block in GeForce 210
#define TxB 512

__global__
void rgba_to_grey_kernel(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
  // The mapping from components of a uchar4 to RGBA is:
  // .x -> R ; .y -> G ; .z -> B ; .w -> A
  //
  //The output (greyImage) at each pixel should be the result of
  //applying the formula: output = .299f * R + .587f * G + .114f * B;
  //Note: We will be ignoring the alpha channel for this conversion
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  uchar4 px = rgbaImage[i]; // thread pixel to process
  greyImage[i] = .299f * px.x +
                 .587f * px.y +
                 .114f * px.z;
}

void rgba_to_grey(uchar4 * const d_rgbaImage,
                  unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{

  // Since it does not matter the relative position of a pixel
  // the block - grid assign strategy will simply be to cover
  // all pixels secuencially in 'x' axis
  long long int total_px = numRows * numCols;  // total pixels
  long int grids_n = ceil(total_px / TxB); // grids numer
  const dim3 blockSize(TxB, 1, 1);
  const dim3 gridSize(grids_n, 1, 1);
  rgba_to_grey_kernel<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}